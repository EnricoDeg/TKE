#include "hip/hip_runtime.h"
#include <iostream>

#include "hip/hip_runtime.h"
#include <cuda/std/mdspan>

#include "utils.hpp"
#include "TKE_cuda.hpp"
#include "cuda_check.hpp"

namespace cudastd = cuda::std;

constexpr auto dyn = cuda::std::dynamic_extent;
using ext1d_t = cuda::std::extents<size_t, dyn>;
using ext2d_t = cuda::std::extents<size_t, dyn, dyn>;
using ext3d_t = cuda::std::extents<size_t, dyn, dyn, dyn>;
using mdspan_1d_double = cudastd::mdspan<double, ext1d_t>;
using mdspan_2d_double = cudastd::mdspan<double, ext2d_t>;
using mdspan_3d_double = cudastd::mdspan<double, ext3d_t>;
using mdspan_2d_int = cudastd::mdspan<int, ext2d_t>;

// this needs to be defined static or TKE_cuda.hpp can not
// be included in C++ files (need to think about other solutions)
static mdspan_1d_double view_cuda_malloc(double *field, size_t dim1);
static mdspan_2d_double view_cuda_malloc(double *field, size_t dim1, size_t dim2);
static mdspan_3d_double view_cuda_malloc(double *field, size_t dim1, size_t dim2, size_t dim3);

// TKE internal memory views
static mdspan_2d_double rho_up_view;
static mdspan_2d_double rho_down_view;
static mdspan_1d_double forc_tke_surf_2D_view;
static mdspan_1d_double forc_rho_surf_2D_view;
static mdspan_1d_double bottom_fric_2D_view;
static mdspan_1d_double s_c_view;
static mdspan_2d_double dzw_stretched_view;
static mdspan_2d_double dzt_stretched_view;
static mdspan_2d_double tke_old_view;
static mdspan_2d_double tke_Av_view;
static mdspan_2d_double tke_kv_view;
static mdspan_2d_double tke_iw_alpha_c_view;
static mdspan_2d_double tke_iwe_view;
static mdspan_2d_double tke_iwe_forcing_view;
static mdspan_2d_double pressure_view;
static mdspan_2d_double Nsqr_view;
static mdspan_2d_double Ssqr_view;

// TKE interface memory views
static mdspan_3d_double tke_view;
static mdspan_2d_int dolic_c_view;

// TKE CUDA kernels functions
__global__ void calc_impl_kernel(int blockNo, int start_index, int end_index,
                                 mdspan_2d_int dolic_c, mdspan_3d_double tke,
                                 mdspan_2d_double tke_old);

TKE_cuda::TKE_cuda(int nproma, int nlevs, int nblocks,
                   int block_size, int start_index, int end_index)
    : TKE_backend(nproma, nlevs, nblocks, block_size, start_index, end_index) {

    // Initialize internal arrays
    std::cout << "Initializing TKE cuda... " << std::endl;
    rho_up_view = view_cuda_malloc(m_rho_up, (size_t)nlevs, (size_t)nproma);
    rho_up_view = view_cuda_malloc(m_rho_down, (size_t)nlevs, (size_t)nproma);
    forc_tke_surf_2D_view = view_cuda_malloc(m_forc_tke_surf_2D, (size_t)nproma);
    forc_rho_surf_2D_view = view_cuda_malloc(m_forc_rho_surf_2D, (size_t)nproma);
    bottom_fric_2D_view = view_cuda_malloc(m_bottom_fric_2D, (size_t)nproma);
    s_c_view = view_cuda_malloc(m_s_c, (size_t)nproma);
    dzw_stretched_view = view_cuda_malloc(m_dzw_stretched, (size_t)(nlevs), (size_t)nproma);
    dzt_stretched_view = view_cuda_malloc(m_dzt_stretched, (size_t)(nlevs+1), (size_t)nproma);
    tke_old_view = view_cuda_malloc(m_tke_old, (size_t)(nlevs+1), (size_t)nproma);
    tke_Av_view = view_cuda_malloc(m_tke_Av, (size_t)(nlevs+1), (size_t)nproma);
    tke_kv_view = view_cuda_malloc(m_tke_kv, (size_t)(nlevs+1), (size_t)nproma);
    tke_iw_alpha_c_view = view_cuda_malloc(m_tke_iw_alpha_c, (size_t)(nlevs+1), (size_t)nproma);
    tke_iwe_view = view_cuda_malloc(m_tke_iwe, (size_t)(nlevs+1), (size_t)nproma);
    tke_iwe_forcing_view = view_cuda_malloc(m_tke_iwe_forcing, (size_t)(nlevs+1), (size_t)nproma);
    pressure_view = view_cuda_malloc(m_pressure, (size_t)(nlevs), (size_t)nproma);
    Nsqr_view = view_cuda_malloc(m_Nsqr, (size_t)(nlevs+1), (size_t)nproma);
    Ssqr_view = view_cuda_malloc(m_Ssqr, (size_t)(nlevs+1), (size_t)nproma);

    is_view_init = false;

}

TKE_cuda::~TKE_cuda() {

    // Free internal arrays memory
    std::cout << "Finalizing TKE cuda... " << std::endl;
    check( hipFree(m_rho_up) );
    check( hipFree(m_rho_down) );
    check( hipFree(m_forc_tke_surf_2D) );
    check( hipFree(m_forc_rho_surf_2D) );
    check( hipFree(m_bottom_fric_2D) );
    check( hipFree(m_s_c) );
    check( hipFree(m_dzw_stretched) );
    check( hipFree(m_dzt_stretched) );
    check( hipFree(m_tke_old) );
    check( hipFree(m_tke_Av) );
    check( hipFree(m_tke_kv) );
    check( hipFree(m_tke_iw_alpha_c) );
    check( hipFree(m_tke_iwe) );
    check( hipFree(m_tke_iwe_forcing) );
    check( hipFree(m_pressure) );
    check( hipFree(m_Nsqr) );
    check( hipFree(m_Ssqr) );

}

void TKE_cuda::calc_impl(int start_block, int end_block, struct t_patch p_patch, struct t_cvmix p_cvmix) {

    if (!is_view_init) {
        tke_view = mdspan_3d_double{ p_cvmix.tke, ext3d_t{m_nblocks,m_nlevs,m_nproma} };
        dolic_c_view = mdspan_2d_int{ p_patch.dolic_c, ext2d_t{m_nblocks,m_nproma} };
        is_view_init = true;
    }

    for (int jb=start_block; jb<=end_block; jb++) {
        int start_index, end_index;
        get_index_range(m_block_size, 0, m_nblocks-1, m_start_index, m_end_index,
                       jb, &start_index, &end_index);
        int threadsPerBlockI = 512;
        int blocksPerGridI = (end_index - start_index) / threadsPerBlockI + 1;
        dim3 blocksPerGrid(blocksPerGridI, 1, 1);
        dim3 threadsPerBlock(threadsPerBlockI, 1, 1);
        calc_impl_kernel<<<blocksPerGrid,threadsPerBlock>>>(jb, start_index, end_index,
                                                            dolic_c_view, tke_view,
                                                            tke_old_view);
    }

}

static mdspan_1d_double view_cuda_malloc(double *field, size_t dim1) {

    check( hipMalloc(&field, dim1*sizeof(double)) );
    mdspan_1d_double memview{ field, ext1d_t{dim1} };
    return memview;

}

static mdspan_2d_double view_cuda_malloc(double *field, size_t dim1, size_t dim2) {

    check( hipMalloc(&field, dim1*dim2*sizeof(double)) );
    mdspan_2d_double memview{ field, ext2d_t{dim1, dim2} };
    return memview;

}

static mdspan_3d_double view_cuda_malloc(double *field, size_t dim1, size_t dim2, size_t dim3) {

    check( hipMalloc(&field, dim1*dim2*dim3*sizeof(double)) );
    mdspan_3d_double memview{ field, ext3d_t{dim1, dim2, dim3} };
    return memview;

}

__global__ void calc_impl_kernel(int blockNo, int start_index, int end_index,
                                 mdspan_2d_int dolic_c, mdspan_3d_double tke,
                                 mdspan_2d_double tke_old) {

    int jc = blockIdx.x * blockDim.x + threadIdx.x + start_index;
    if (jc <= end_index) {
        int levels = dolic_c(blockNo,jc);
        for (int level = 0; level < levels; level++) {
            tke_old(level,jc) = tke(blockNo,level,jc);
            tke(blockNo,level,jc) = tke(blockNo,level,jc) + 1.0;
        }
    }

}
