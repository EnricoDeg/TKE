/* Copyright (C) 2023  Enrico Degregori, Wilton Jaciel Loch
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include <algorithm>
#include <iostream>
#include "src/TKE_cuda.hpp"
#include "src/utils.hpp"
#include "src/cuda_kernels.hpp"

// Structures with memory views
struct t_cvmix_view p_cvmix_view_l;
struct t_patch_view p_patch_view_l;
struct t_ocean_state_view ocean_state_view_l;
struct t_atmo_fluxes_view atmos_fluxes_view_l;
struct t_atmos_for_ocean_view p_as_view_l;
struct t_sea_ice_view p_sea_ice_view_l;
struct t_tke_internal_view p_internal_view_l;

// Structures with parameters
struct t_constant p_constant;
struct t_constant_tke p_constant_tke;

TKE_cuda::TKE_cuda(int nproma, int nlevs, int nblocks, int vert_mix_type, int vmix_idemix_tke,
                   int vert_cor_type, double dtime, double OceanReferenceDensity, double grav,
                   int l_lc, double clc, double ReferencePressureIndbars, double pi)
    : TKE_backend(nproma, nlevs, nblocks, vert_mix_type, vmix_idemix_tke,
                  vert_cor_type, dtime, OceanReferenceDensity, grav,
                  l_lc, clc, ReferencePressureIndbars, pi) {
    // Allocate internal arrays memory and create memory views
    std::cout << "Initializing TKE cuda... " << std::endl;
    p_internal_view_l.tke_old = view_cuda_malloc(m_tke_old, static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.rho_up = view_cuda_malloc(m_rho_up, static_cast<size_t>(nlevs), static_cast<size_t>(nproma));
    p_internal_view_l.rho_down = view_cuda_malloc(m_rho_down, static_cast<size_t>(nlevs), static_cast<size_t>(nproma));
    p_internal_view_l.forc_tke_surf_2D = view_cuda_malloc(m_forc_tke_surf_2D, static_cast<size_t>(nproma));
    p_internal_view_l.forc_rho_surf_2D = view_cuda_malloc(m_forc_rho_surf_2D, static_cast<size_t>(nproma));
    p_internal_view_l.bottom_fric_2D = view_cuda_malloc(m_bottom_fric_2D, static_cast<size_t>(nproma));
    p_internal_view_l.s_c = view_cuda_malloc(m_s_c, static_cast<size_t>(nproma));
    p_internal_view_l.dzw_stretched = view_cuda_malloc(m_dzw_stretched,
                                                       static_cast<size_t>(nlevs), static_cast<size_t>(nproma));
    p_internal_view_l.dzt_stretched = view_cuda_malloc(m_dzt_stretched,
                                                       static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.tke_Av = view_cuda_malloc(m_tke_Av,
                               static_cast<size_t>(nblocks), static_cast<size_t>(nlevs+1),
                               static_cast<size_t>(nproma));
    p_internal_view_l.tke_kv = view_cuda_malloc(m_tke_kv, static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.tke_iw_alpha_c = view_cuda_malloc(m_tke_iw_alpha_c,
                                                        static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.tke_iwe = view_cuda_malloc(m_tke_iwe, static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.tke_iwe_forcing = view_cuda_malloc(m_tke_iwe_forcing, static_cast<size_t>(nlevs+1),
                                            static_cast<size_t>(nproma));
    p_internal_view_l.pressure = view_cuda_malloc(m_pressure, static_cast<size_t>(nlevs), static_cast<size_t>(nproma));
    p_internal_view_l.Nsqr = view_cuda_malloc(m_Nsqr, static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.Ssqr = view_cuda_malloc(m_Ssqr, static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.a_dif = view_cuda_malloc(m_a_dif, static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.b_dif = view_cuda_malloc(m_b_dif, static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.c_dif = view_cuda_malloc(m_c_dif, static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.a_tri = view_cuda_malloc(m_a_tri, static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.b_tri = view_cuda_malloc(m_b_tri, static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.c_tri = view_cuda_malloc(m_c_tri, static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.d_tri = view_cuda_malloc(m_d_tri, static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.mxl = view_cuda_malloc(m_mxl, static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.sqrttke = view_cuda_malloc(m_sqrttke, static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.KappaM_out = view_cuda_malloc(m_KappaM_out,
                                                    static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.Rinum = view_cuda_malloc(m_Rinum, static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.prandtl = view_cuda_malloc(m_prandtl, static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.KappaH_out = view_cuda_malloc(m_KappaH_out,
                                                    static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.forc = view_cuda_malloc(m_forc, static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.K_diss_v = view_cuda_malloc(m_K_diss_v,
                                                  static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.P_diss_v = view_cuda_malloc(m_P_diss_v,
                                                  static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.ke = view_cuda_malloc(m_ke, static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.cp = view_cuda_malloc(m_cp, static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.dp = view_cuda_malloc(m_dp, static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.tke_upd = view_cuda_malloc(m_tke_upd,
                                                 static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    p_internal_view_l.tke_unrest = view_cuda_malloc(m_tke_unrest,
                                                    static_cast<size_t>(nlevs+1), static_cast<size_t>(nproma));
    is_view_init = false;
}

TKE_cuda::~TKE_cuda() {
    // Free internal arrays memory
    std::cout << "Finalizing TKE cuda... " << std::endl;
    check(hipFree(m_rho_up));
    check(hipFree(m_rho_down));
    check(hipFree(m_forc_tke_surf_2D));
    check(hipFree(m_forc_rho_surf_2D));
    check(hipFree(m_bottom_fric_2D));
    check(hipFree(m_s_c));
    check(hipFree(m_dzw_stretched));
    check(hipFree(m_dzt_stretched));
    check(hipFree(m_tke_old));
    check(hipFree(m_tke_Av));
    check(hipFree(m_tke_kv));
    check(hipFree(m_tke_iw_alpha_c));
    check(hipFree(m_tke_iwe));
    check(hipFree(m_tke_iwe_forcing));
    check(hipFree(m_pressure));
    check(hipFree(m_Nsqr));
    check(hipFree(m_Ssqr));
    check(hipFree(m_a_dif));
    check(hipFree(m_b_dif));
    check(hipFree(m_c_dif));
    check(hipFree(m_a_tri));
    check(hipFree(m_b_tri));
    check(hipFree(m_c_tri));
    check(hipFree(m_d_tri));
    check(hipFree(m_mxl));
    check(hipFree(m_sqrttke));
    check(hipFree(m_KappaM_out));
    check(hipFree(m_Rinum));
    check(hipFree(m_prandtl));
    check(hipFree(m_KappaH_out));
    check(hipFree(m_forc));
    check(hipFree(m_ke));
    check(hipFree(m_cp));
    check(hipFree(m_dp));
    check(hipFree(m_tke_upd));
    check(hipFree(m_tke_unrest));
}

void TKE_cuda::calc_impl(t_patch p_patch, t_cvmix p_cvmix,
                         t_ocean_state ocean_state, t_atmo_fluxes atmos_fluxes,
                         t_atmos_for_ocean p_as, t_sea_ice p_sea_ice,
                         int edges_block_size, int edges_start_block, int edges_end_block,
                         int edges_start_index, int edges_end_index, int cells_block_size,
                         int cells_start_block, int cells_end_block, int cells_start_index,
                         int cells_end_index) {
    if (!is_view_init) {
        fill_struct_view(&p_cvmix_view_l, &p_cvmix, m_nblocks, m_nlevs, m_nproma);
        fill_struct_view(&p_patch_view_l, &p_patch, m_nblocks, m_nlevs, m_nproma);
        fill_struct_view(&ocean_state_view_l, &ocean_state, m_nblocks, m_nlevs, m_nproma);
        fill_struct_view(&atmos_fluxes_view_l, &atmos_fluxes, m_nblocks, m_nlevs, m_nproma);
        fill_struct_view(&p_as_view_l, &p_as, m_nblocks, m_nlevs, m_nproma);
        fill_struct_view(&p_sea_ice_view_l, &p_sea_ice, m_nblocks, m_nlevs, m_nproma);

        p_constant.vert_mix_type = m_vert_mix_type;
        p_constant.vmix_idemix_tke = m_vmix_idemix_tke;
        p_constant.vert_cor_type = m_vert_cor_type;
        p_constant.dtime = m_dtime;
        p_constant.OceanReferenceDensity = m_OceanReferenceDensity;
        p_constant.grav = m_grav;
        p_constant.l_lc = m_l_lc;
        p_constant.clc = m_clc;
        p_constant.ReferencePressureIndbars = m_ReferencePressureIndbars;
        p_constant.pi = m_pi;
        p_constant.nlevs = m_nlevs;

        p_constant_tke.c_k = 0.1;
        p_constant_tke.c_eps = 0.7;
        p_constant_tke.cd = 3.75;
        p_constant_tke.alpha_tke = 30.0;
        p_constant_tke.clc = 0.15;
        p_constant_tke.mxl_min = 1.0e-8;
        p_constant_tke.KappaM_min = 1.0e-4;
        p_constant_tke.KappaH_min = 1.0e-5;
        p_constant_tke.KappaM_max = 100.0;
        p_constant_tke.tke_surf_min = 1.0e-4;
        p_constant_tke.tke_min = 1.0e-6;
        p_constant_tke.tke_mxl_choice = 2;
        p_constant_tke.handle_old_vals = 1;
        p_constant_tke.only_tke = true;
        p_constant_tke.use_Kappa_min = false;
        p_constant_tke.use_ubound_dirichlet = false;
        p_constant_tke.use_lbound_dirichlet = false;

        is_view_init = true;
    }

    for (int jb = cells_start_block; jb <= cells_end_block; jb++) {
        int start_index, end_index;
        get_index_range(cells_block_size, cells_start_block, cells_end_block,
                        cells_start_index, cells_end_index, jb, &start_index, &end_index);
        int threadsPerBlockI = 512;
        int blocksPerGridI = (end_index - start_index) / threadsPerBlockI + 1;
        dim3 blocksPerGrid(blocksPerGridI, 1, 1);
        dim3 threadsPerBlock(threadsPerBlockI, 1, 1);
        calc_impl_cells<<<blocksPerGrid, threadsPerBlock>>>(jb, start_index, end_index,
                                                            p_patch_view_l, p_cvmix_view_l,
                                                            ocean_state_view_l, atmos_fluxes_view_l,
                                                            p_as_view_l, p_sea_ice_view_l,
                                                            p_internal_view_l, p_constant,
                                                            p_constant_tke);
    }
/*
    for (int jb = edges_start_block; jb <= edges_end_block; jb++) {
        int start_index, end_index;
        get_index_range(edges_block_size, edges_start_block, edges_end_block,
                        edges_start_index, edges_end_index, jb, &start_index, &end_index);
        int threadsPerBlockI = 512;
        int blocksPerGridI = (end_index - start_index) / threadsPerBlockI + 1;
        dim3 blocksPerGrid(blocksPerGridI, 1, 1);
        dim3 threadsPerBlock(threadsPerBlockI, 1, 1);
        calc_impl_edges<<<blocksPerGrid, threadsPerBlock>>>(jb, start_index, end_index,
                                                            p_patch_view_l, p_cvmix_view_l,
                                                            p_internal_view_l, p_constant);
    }
*/
    check(hipDeviceSynchronize());
}
