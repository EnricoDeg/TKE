#include <iostream>
#include "hip/hip_runtime.h"
#include <cuda/std/mdspan>
#include "TKE_cuda.hpp"

namespace stdex = cuda::std;

TKE_cuda::TKE_cuda(int nproma, int nlevs, int nblocks)
    : TKE_backend(nproma, nlevs, nblocks) {
    hipMalloc(&rho_up, nproma*nlevs*sizeof(double));
}

TKE_cuda::~TKE_cuda() {
    std::cout << "Finalizing TKE cuda... " << std::endl;
    hipFree(rho_up);
}

void TKE_cuda::calc_impl() {

}
