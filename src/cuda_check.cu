#include <iostream>
#include "cuda_check.hpp"

void check(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        std::exit(-1);
    }
}
